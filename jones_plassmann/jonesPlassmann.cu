#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>
#include <cstdlib>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <thrust/count.h>

#define THREADS_PER_BLOCK 256

void check_error(std::string s) {
    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "%s: WARNING: A CUDA error occured: code=%d, %s\n", s.c_str(), errCode,
                hipGetErrorString(errCode));
        exit(1);
    }
}

// assign random numbers to each node
int* init_rank(int node_cnt) {
    int* rank = (int*) malloc(sizeof(int) * node_cnt);
    for (int i = 0; i < node_cnt; ++i) {
        rank[i] = rand();
    }
    return rank;
}

__global__ void jones_plassmann_kernel(int cur_color, int node_cnt, 
    int* colors, int *nbrs_start, int *nbrs, int* rank) {
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    // already colored, skip
    if (node >= node_cnt || colors[node] != 0) return;
    
    bool is_max = true;
    for (int nbr_idx = nbrs_start[node]; nbr_idx < nbrs_start[node + 1]; ++nbr_idx) {
        int nbr = nbrs[nbr_idx];
        // ignore colored neighbor
        if (colors[nbr] != 0) {
            continue;
        }
        if (rank[node] <= rank[nbr]) is_max = false;
    }
    if (is_max) colors[node] = cur_color;
}

void jones_plassmann(int node_cnt, int edge_cnt, int* colors, int *nbrs_start, int *nbrs) {
    // initialization
    int num_blocks = (node_cnt + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int* rank = init_rank(node_cnt);

    // allcoate memory
    int* device_nbrs;
    int* device_nbrs_start;
    int* device_colors;
    int* device_rank;
    hipMalloc(&device_nbrs, edge_cnt * 2);
    hipMalloc(&device_nbrs_start, node_cnt + 1);
    hipMalloc(&device_colors, node_cnt);
    hipMalloc(&device_rank, node_cnt);
    // copy input to device
    hipMemcpy(device_rank, rank, node_cnt, hipMemcpyHostToDevice);
    hipMemcpy(device_colors, colors, node_cnt, hipMemcpyHostToDevice);
    hipMemcpy(device_nbrs, nbrs, edge_cnt * 2, hipMemcpyHostToDevice);
    hipMemcpy(device_nbrs_start, nbrs_start, node_cnt + 1, hipMemcpyHostToDevice);

    for (int cur_color = 1; cur_color <= node_cnt; ++cur_color) {
        jones_plassmann_kernel<<<num_blocks, THREADS_PER_BLOCK>>>
            (cur_color, node_cnt, device_colors, device_nbrs_start, device_nbrs, device_rank);
        hipMemcpy(colors, device_colors, node_cnt, hipMemcpyDeviceToHost);
        int uncolored = (int)thrust::count(colors, colors + node_cnt, 0);
        if (uncolored == 0) break;
    }
    
    // free memory
    hipFree(device_rank);
    hipFree(device_nbrs);
    hipFree(device_nbrs_start);
    hipFree(device_colors);
}

void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}