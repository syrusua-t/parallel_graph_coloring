#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include <thrust/count.h>

#define THREADS_PER_BLOCK 256

// assign random numbers to each node
int* init_rank(int node_cnt) {
    // TODO
    return nullptr;
}

__global__ void jones_plassmann_kernel() {
    // TODO
}

void jones_plassmann(int* colors, const std::vector<std::vector<int>>& graph) {
    int node_cnt = graph.size();
    int num_blocks = (node_cnt + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int* rank = init_rank(node_cnt);

    for (int i = 0; i < node_cnt; ++i) {
        jones_plassmann_kernel<<<num_blocks, THREADS_PER_BLOCK>>>();
        int uncolored = (int)thrust::count(colors, colors + node_cnt, 0);
        if (uncolored == 0) break;
    }
}


void printCudaInfo() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}